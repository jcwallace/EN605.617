#include "project.h"

#include <hipblas.h>

void cublas(float* A, float* B, float* C,
               int K, int M, int N,
               int timerID) {

    // cuBLAS configuration
    hipblasStatus_t status;
    hipblasHandle_t handle;
    status = hipblasCreate(&handle);

    // Prepare CUDA memory objects
    float* bufA = 0;
    float* bufB = 0;
    float* bufC = 0;
    hipMalloc((void**)&bufA, M*K*sizeof(*A));
    hipMalloc((void**)&bufB, K*N*sizeof(*B));
    hipMalloc((void**)&bufC, M*N*sizeof(*C));

    // Copy matrices to the GPU (also C to erase the results of the previous run)
    hipMemcpy((void*)bufA, (void*)A, M*K*sizeof(*A), hipMemcpyHostToDevice);
    hipMemcpy((void*)bufB, (void*)B, K*N*sizeof(*B), hipMemcpyHostToDevice);
    hipMemcpy((void*)bufC, (void*)C, M*N*sizeof(*C), hipMemcpyHostToDevice);

    // Configure SGEMM
    float alpha = ALPHA;
    float beta = BETA;

    // Start the timed loop
    double startTime = timer();
    for (int r=0; r<NUM_RUNS; r++) {

        // Call cuBLAS
        status = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                             M, N, K, &alpha,
                             bufA, M,
                             bufB, K, &beta,
                             bufC, M);

        // Wait for calculations to be finished
        hipDeviceSynchronize();
    }

    // End the timed loop
    timers[timerID].t += (timer() - startTime) / (double)NUM_RUNS;
    timers[timerID].kf += ((long)K * (long)M * (long)N * 2) / 1000;

    // Copy the output matrix C back to the CPU memory
    hipMemcpy((void*)C, (void*)bufC, M*N*sizeof(*C), hipMemcpyDeviceToHost);

    // Free the GPU memory objects
    hipFree(bufA);
    hipFree(bufB);
    hipFree(bufC);

    // Clean-up cuBLAS
    status = hipblasDestroy(handle);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        exit(1);
    }
}