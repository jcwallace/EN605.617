//Based on the work of Andrew Krepps
#include <stdio.h>
#include <stdlib.h>
#include <chrono>
#include <assert.h>
    
// CUDA Libraries
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsolver.h>
    
#define MAX 50   
#define index(i,j,ld) (((j)*(ld))+(i))

__global__
void init(unsigned int seed, hiprandState_t* states) {
    const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;  
    hiprand_init(seed, thread_idx, 0, &states[thread_idx]);
}
 
__global__
void random(hiprandState_t* states, float* numbers) {
  const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
  numbers[thread_idx] = hiprand(&states[thread_idx]) % MAX;
}
    
void printMatrix(float*P,int n)
{
    if( n > 10 ){
        printf("\nOmitting (too large)...\n");
        return;
    }
    
    int i,j;
    for(i=0;i<n;i++){
        printf("\n");
        for(j=0;j<n;j++)
            printf("%f ",P[index(i,j,n)]);
    }
    printf("\n");
}

void printMatrix(int n, double*P)
{
  //printf("\n %f",P[1]);
  int i,j;
  for(i=0;i<n;i++){

      printf("\n");

      for(j=0;j<n;j++)
          printf("%f ",P[index(i,j,n)]);
  }
  printf("\n");
}
    
void main_MM(int mDim){
    int matrixDim = mDim;
    int mSize = mDim*mDim;
    // ---------------------------------- //
    // ----  Random Matrix Generate  ---- //
    // ---------------------------------- //
    float *a_host = (float*)malloc(mSize*sizeof(float));
    float *b_host = (float*)malloc(mSize*sizeof(float));
    float *c_host = (float*)malloc(mSize*sizeof(float));
    float *A, *B, *C;
    hiprandState_t* states1;
    hiprandState_t* states2;
        
    hipMalloc((void**) &states1, mSize * sizeof(hiprandState_t));
    init<<<matrixDim, matrixDim>>>(42, states1);
    hipMalloc((void**) &states2, mSize * sizeof(hiprandState_t));
    init<<<matrixDim, matrixDim>>>(time(0), states2);
    
    hipMalloc((void**) &A, sizeof(float)*mSize );
    hipMalloc((void**) &B, sizeof(float)*mSize );
    hipMalloc((void**) &C, sizeof(float)*mSize );
    
    random<<<matrixDim, matrixDim>>>(states1, A);
    random<<<matrixDim, matrixDim>>>(states2, B);
    
    hipMemcpy(a_host, A, mSize * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(b_host, B, mSize * sizeof(float), hipMemcpyDeviceToHost);
    
    printf("___A___");
    printMatrix(a_host,matrixDim);
    
    printf("___B___");
    printMatrix(b_host,matrixDim);
    
    // ----------------------- //
    // ----  Matrix Mult  ---- //
    // ----------------------- //
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasStatus_t ret;
    const float alpha = 1.0f;
    const float beta  = 0.0f;
    auto start = std::chrono::high_resolution_clock::now();
    ret = hipblasSgemm
                (
                    handle,
                    HIPBLAS_OP_N,
                    HIPBLAS_OP_N,
                    matrixDim,
                    matrixDim,
                    matrixDim,
                    &alpha,
                    A,
                    matrixDim,
                    B,
                    matrixDim,
                    &beta,
                    C,
                    matrixDim
                );
    
    if (ret != HIPBLAS_STATUS_SUCCESS)
    {
        printf("hipblasSgemm returned error code %d, line(%d)\n", ret, __LINE__);
    }
    auto stop = std::chrono::high_resolution_clock::now(); 
    float delta = std::chrono::duration<double,std::milli>(stop - start).count();
    printf("-- Matrix Multiplication (%dx%d) [%f ms]\n", mDim, mDim, delta);
    
    hipMemcpy(c_host, C, mSize * sizeof(float), hipMemcpyDeviceToHost);
    
    printf("___A*B___ ");
    printMatrix(c_host,matrixDim);
    
    free( a_host );  free( b_host );  free ( c_host );
    hipFree(A);
    hipFree(B);
    hipFree(C);
    hipDeviceReset();
}
    
void main_eigenSolve()
{
    // ------------------------ //
    // ----  Eigen Solver  ---- //
    // ------------------------ //
    printf("\nSolving for Eiganvalues");
    hipsolverHandle_t cusolverH = NULL;
    hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;
    hipError_t cudaStat1 = hipSuccess;
    hipError_t cudaStat2 = hipSuccess;
    hipError_t cudaStat3 = hipSuccess;
    const int m = 3;
    const int lda = m;

    
    double A[lda*m] = { 3.5, 0.5, 0, 0.5, 3.5, 0, 0, 0, 2.0};
    double lambda[m] = { 2.0, 3.0, 4.0};
    double V[lda*m]; // eigenvectors
    double W[m]; // eigenvalues
    double *d_A = NULL;
    double *d_W = NULL;
    int *devInfo = NULL;
    double *d_work = NULL;
    int lwork = 0;
    int info_gpu = 0;

    cusolver_status = hipsolverDnCreate(&cusolverH);
    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
    
    cudaStat1 = hipMalloc ((void**)&d_A, sizeof(double) * lda * m);
    cudaStat2 = hipMalloc ((void**)&d_W, sizeof(double) * m);
    cudaStat3 = hipMalloc ((void**)&devInfo, sizeof(int));
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);
    cudaStat1 = hipMemcpy(d_A, A, sizeof(double) * lda * m, hipMemcpyHostToDevice);
    assert(hipSuccess == cudaStat1);

    // query working space of syevd
    hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR; // compute eigenvalues and eigenvectors.
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
    cusolver_status = hipsolverDnDsyevd_bufferSize
                        (
                            cusolverH,
                            jobz,
                            uplo,
                            m,
                            d_A,
                            lda,
                            d_W,
                            &lwork
                        );
    assert (cusolver_status == HIPSOLVER_STATUS_SUCCESS);
    cudaStat1 = hipMalloc((void**)&d_work, sizeof(double)*lwork);
    assert(hipSuccess == cudaStat1);

    // compute spectrum
    cusolver_status = hipsolverDnDsyevd
                        (
                            cusolverH,
                            jobz,
                            uplo,
                            m,
                            d_A,
                            lda,
                            d_W,
                            d_work,
                            lwork,
                            devInfo
                       );
    cudaStat1 = hipDeviceSynchronize();
    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
    assert(hipSuccess == cudaStat1);
    
    cudaStat1 = hipMemcpy(W, d_W, sizeof(double)*m, hipMemcpyDeviceToHost);
    cudaStat2 = hipMemcpy(V, d_A, sizeof(double)*lda*m, hipMemcpyDeviceToHost);
    cudaStat3 = hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);
    
    printf("eigenvalue(s) = \n");
    for(int i = 0 ; i < m ; i++){
        printf("W[%d] = %E\n", i+1, W[i]);
    }
                         
       
    // check eigenvalues
    double lambda_sup = 0;
    for(int i = 0 ; i < m ; i++){
        double error = fabs( lambda[i] - W[i]);
        lambda_sup = (lambda_sup > error)? lambda_sup : error;
    }
    printf("|lambda - W| = %E\n", lambda_sup);
    
    // free resources
    if (d_A ) hipFree(d_A);
    if (d_W ) hipFree(d_W);
    if (devInfo) hipFree(devInfo);
    if (d_work ) hipFree(d_work);
    if (cusolverH) hipsolverDnDestroy(cusolverH);
    hipDeviceReset();
}
    
    
int main(int argc, char** argv)
{
	// read command line arguments
	int matrixDim = 3;
    	
	if (argc >= 2) {
		matrixDim = atoi(argv[1]);
	}
	// validate command line arguments
	if (matrixDim == 0) {
		matrixDim = 3;
		
		printf("Warning: Size specified too small (using default 3)\n");
	}
    
    printf("Matrix Dim: %d\n",matrixDim);
    
    main_MM(matrixDim);
    main_MM(matrixDim*10);
    main_MM(matrixDim*100);
    
    main_eigenSolve();
    
}
