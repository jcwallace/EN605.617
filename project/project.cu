//Based on the work of Andrew Krepps

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <chrono>
    
#define NUM_ELEMENTS 100000    

__global__
void add(int * arr1, int * arr2, int * out)
{    
    const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    out[thread_idx] = arr1[thread_idx] + arr2[thread_idx];
}
    

__global__  
void sub(int * arr1, int * arr2, int * out){
    const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    out[thread_idx] = arr1[thread_idx] - arr2[thread_idx];
}
    

__global__
void mult(int * arr1, int * arr2, int * out){
    const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    out[thread_idx] = arr1[thread_idx] * arr2[thread_idx];
}
    

__global__ 
void mod(int * arr1, int * arr2, int * out){
    const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    out[thread_idx] = arr1[thread_idx] % arr2[thread_idx];
}

       
void main_math(int blockSize, int numBlocks) {
    
    // Main Assignment Function calls
    int array1[NUM_ELEMENTS];
    int array2[NUM_ELEMENTS];
    int output[NUM_ELEMENTS];
    
    for( int i = 0; i<NUM_ELEMENTS; i++){
        array1[i] = i;
        array2[i] = rand() % 4;
    }
    
    int *arr1,*arr2,*out;
    
    int arraySize_bytes = (sizeof(unsigned int) * NUM_ELEMENTS);                             
    hipMalloc((void **)&arr1, arraySize_bytes);
    hipMalloc((void **)&arr2, arraySize_bytes);
    hipMalloc((void **)&out, arraySize_bytes);
                   
    auto start = std::chrono::high_resolution_clock::now();                                
    hipMemcpy(arr1,array1,arraySize_bytes,hipMemcpyHostToDevice);                                
    hipMemcpy(arr2,array2,arraySize_bytes,hipMemcpyHostToDevice);                                 
                                 
    add<<<numBlocks,blockSize>>>(arr1,arr2,out);
    sub<<<numBlocks,blockSize>>>(arr1,arr2,out);
    mult<<<numBlocks,blockSize>>>(arr1,arr2,out);
    mod<<<numBlocks,blockSize>>>(arr1,arr2,out);
    
    hipMemcpy(output,out,arraySize_bytes,hipMemcpyDeviceToHost);
    
    auto stop = std::chrono::high_resolution_clock::now(); 
    float delta = std::chrono::duration<double,std::milli>(stop - start).count();
    printf("-- Global memory (Baseline) [%f ms]\n", delta);
    
    hipFree(arr1);
    hipFree(arr2);
    hipFree(out);
    
}

 
void main_math_stream(int blockSize, int numBlocks){
    
    int arraySize_bytes = (sizeof(unsigned int) * NUM_ELEMENTS);
    int *host_arr1, *host_arr2, *host_out;
    int *dev_arr1, *dev_arr2, *dev_out;
                                    
    hipStream_t stream; 
    hipStreamCreate(&stream);
                                                                    
    hipMalloc((void **)&dev_arr1, arraySize_bytes);
    hipMalloc((void **)&dev_arr2, arraySize_bytes);
    hipMalloc((void **)&dev_out, arraySize_bytes);
    
    hipHostAlloc((void **)&host_arr1, arraySize_bytes, hipHostMallocDefault);
    hipHostAlloc((void **)&host_arr2, arraySize_bytes, hipHostMallocDefault);
    hipHostAlloc((void **)&host_out, arraySize_bytes, hipHostMallocDefault);
    
    for( int i = 0; i<NUM_ELEMENTS; i++){
        host_arr1[i] = i;
        host_arr2[i] = rand() % 4;
    }
        
    auto start = std::chrono::high_resolution_clock::now();                                
                                    
    hipMemcpyAsync(dev_arr1, host_arr1, arraySize_bytes, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(dev_arr2, host_arr2, arraySize_bytes, hipMemcpyHostToDevice, stream);                                
                                    
    add<<<NUM_ELEMENTS,numBlocks,blockSize,stream>>>(dev_arr1,dev_arr2,dev_out);
    sub<<<NUM_ELEMENTS,numBlocks,blockSize,stream>>>(dev_arr1,dev_arr2,dev_out);
    mult<<<NUM_ELEMENTS,numBlocks,blockSize,stream>>>(dev_arr1,dev_arr2,dev_out);
    mod<<<NUM_ELEMENTS,numBlocks,blockSize,stream>>>(dev_arr1,dev_arr2,dev_out);
    
    hipMemcpyAsync(host_out, dev_out, arraySize_bytes, hipMemcpyDeviceToHost, stream);
    hipStreamSynchronize(stream);
    
    auto stop = std::chrono::high_resolution_clock::now(); 
    float delta = std::chrono::duration<double,std::milli>(stop - start).count();
    printf("-- Stream and Event [%f ms]\n", delta);
    
    hipHostFree(host_arr1);
    hipHostFree(host_arr2);
    hipHostFree(host_out);
    hipFree(dev_arr1);
    hipFree(dev_arr2);
    hipFree(dev_out);
    hipDeviceReset();
}
    
    
int main(int argc, char** argv)
{
	// read command line arguments
	int totalThreads = 256;
	int blockSize = 256;
    int numBlocks = totalThreads/blockSize;
    	
	if (argc >= 2) {
		totalThreads = atoi(argv[1]);
	}
	if (argc >= 3) {
		blockSize = atoi(argv[2]);
	}

	// validate command line arguments
	if (totalThreads % blockSize != 0) {
		++numBlocks;
		totalThreads = numBlocks*blockSize;
		
		printf("Warning: Total thread count is not evenly divisible by the block size\n");
		printf("The total number of threads will be rounded up to %d\n", totalThreads);
	}
    
    numBlocks = totalThreads/blockSize;
    
    printf("Thread Count: %d\n",totalThreads);
    printf("Block Size: %d\n",blockSize);
    printf("Array Size: %d\n",NUM_ELEMENTS);
    
    main_math(blockSize, numBlocks);
    main_math_stream(blockSize, numBlocks);
    
}
